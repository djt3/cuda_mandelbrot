
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

#define ITTERATIONS 10000

__global__ void run(float* array, int max, double zoom, double x_target, double y_target) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	zoom = zoom * zoom;

	double norm_x = ((double)x / (max)) * (4 / zoom) - 2 / zoom + x_target;
	double norm_y = ((double)y / (max)) * (4 / zoom) - 2 / zoom + y_target;

	double x1 = 0, y1 = 0;

	bool diverging = false;

	for (int i = 0; i < ITTERATIONS; i++) {
		if (sqrtf(x1 * x1 + y1 * y1) >= 2.f) {
			array[x + y * max] = (i / (float)ITTERATIONS) * ((float)ITTERATIONS / 100);
			diverging = true;
			break;
		}

		double xx = (x1 * x1) - (y1 * y1) + norm_x;
		y1 = 2 * x1 * y1 + norm_y;
		x1 = xx;
	}
	if (!diverging)
		array[x + y * max] = 0.f;
}

float* arr;

void generate_image(int max, double zoom, double x_target, double y_target) {
	hipMallocManaged(&arr, max * max * sizeof(float));

	run<<<dim3 (128, 128), dim3 (max / 128, max / 128)>>>(arr, max, zoom, x_target, y_target);
	hipDeviceSynchronize();
}

float get_mandelbrot_at_point(int x, int y, int max) {
	return arr[x + y * max];
}